
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

#define CUDA_TREADS_CU 1
// Kernel function to add the elements of two arrays
	__global__
void add(int n, float *x, float *y)
{
	for (int i = threadIdx.x; i < n; i += CUDA_TREADS_CU)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1<<20;
	float *x = (float*) malloc(N*sizeof(float));
	float *y = (float*) malloc(N*sizeof(float));
	 
	float *dx, *dy;
	// Allocate Unified Memory – accessible from CPU or GPU
	hipMalloc(&dx, N*sizeof(float));
	hipMalloc(&dy, N*sizeof(float));
	//
	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = (float)i;
	}
	hipMemcpy(dx, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dy, y, N*sizeof(float), hipMemcpyHostToDevice);
	// Run kernel on 1M elements on the GPU
	add<<<1, CUDA_TREADS_CU>>>(N, dx, dy);

	// Wait for GPU to finish before accessing on host
	hipMemcpy(x, dx, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y, dy, N*sizeof(float), hipMemcpyDeviceToHost);

	// Check for errors (all values should be 3.0f)

	std::cout<< y[N-1] << std::endl;


	// Free memory
	hipFree(dx);
	hipFree(dy);
	free(x);
	free(y);

	printf("aaa\n");
	return 0;
}
